#include "hip/hip_runtime.h"
#include "cu_cvf.h"
#include "hip/hip_runtime.h"
#include ""
#include <math.h>

#define DIM 32

__host__ __device__ __forceinline__ 
int divUp(int total, int grain)		// ����ȡ������
{
	return (total + grain - 1) / grain;
}

__host__ __device__ __forceinline__ 
unsigned char saturate_cast(int v) // ת��Ϊ0~255
{
	unsigned char r;
	if (v > 255){ r = 255; }
	else if (v < 0){ r = 0; }
	else { r = (unsigned char)v; }
	return r;
}

__host__ __device__ __forceinline__
int clip(int x, int v_min, int v_max)
{
	int r = 0;
	r = (x >= v_min) ? x : v_min;
	r = (r <= v_max) ? r : v_max;
	return r;
}

__device__ __forceinline__ 
float norm_l1(const float& a)  { return fabs(a); }


void CuBilateralFilter::init(int _img_width, int _img_heigth, int _img_channels)
{
	this->img_cols = _img_width;
	this->img_rows = _img_heigth;
	this->img_channels = _img_channels;
	this->img_size = _img_width * _img_heigth * _img_channels;
	hipMalloc( (void**)&dev_input_img, this->img_size * sizeof(unsigned char) );
    hipMalloc( (void**)&dev_result_img, this->img_size * sizeof(unsigned char) );
}

void CuBilateralFilter::release()
{
	hipFree( dev_input_img );
    hipFree( dev_result_img );
}

__global__ 
void bilateral_kernel(unsigned char *src, unsigned char *dst,
					int cols, int rows, const int ksz, const float sigma_spatial2_inv_half, const float sigma_color2_inv_half)
{
	int x = threadIdx.x + blockIdx.x * blockDim.x;
	int y = threadIdx.y + blockIdx.y * blockDim.y;
	if (x >= cols || y >= rows)
		return;
	int r = ksz / 2;
	float r2 = (float)(r * r);
	int offset = y * cols + x;

	//bgr
	float center[3] = { src[offset], src[cols * rows + offset], src[2 * cols * rows + offset] };
	float value[3] = { 0, 0, 0 };
	float weight[3] = { 0, 0, 0 };
	float sum1[3] = {0,0,0};
	float sum2[3] = { 0, 0, 0 };

	int x_tmp, y_tmp, tmp_offset;
	float space2;
	if (x >= 0 && y >= 0 && x < cols && y < rows)
	{
		for (int cy = -r; cy <= r; cy++)
		{
			for (int cx = -r; cx <=r; cx++)
			{
				space2 = cx*cx + cy*cy;
				if (space2 > r2) continue;
				x_tmp = clip(x + cx, 0, cols - 1);
				y_tmp = clip(y + cy, 0, rows - 1);
				tmp_offset = y_tmp * cols + x_tmp;
				for (int k = 0; k < 3; k++)
				{
					value[k] = (float)src[tmp_offset + cols * rows * k];
					weight[k] = expf(space2 * sigma_spatial2_inv_half + sqrtf(norm_l1(value[k] - center[k])) * sigma_color2_inv_half);
					sum1[k] = sum1[k] + weight[k] * value[k];
					sum2[k] = sum2[k] + weight[k];
				}
			}
		}
	}

	//__syncthreads();
	for (int k = 0; k < 3; k++)
	{
		dst[offset + cols * rows * k] = saturate_cast((int)(sum1[k] / sum2[k]));
	}
}

void CuBilateralFilter::bilateral_caller(unsigned char *host_src_img, unsigned char *host_dst_img, int kernel_size, float sigma_spatial, float sigma_color)
{
	dim3 block(32, 32);
	dim3 grid(divUp(this->img_cols, block.x), divUp(this->img_rows, block.y));

	float sigma_spatial2_inv_half = -0.5f / (sigma_spatial * sigma_spatial);
	float sigma_color2_inv_half = -0.5f / (sigma_color * sigma_color);

	hipMemcpy(dev_input_img, host_src_img,
			this->img_size * sizeof(unsigned char), hipMemcpyHostToDevice );
	
	//hipDeviceSynchronize();
	bilateral_kernel << <grid, block >> >(dev_input_img, dev_result_img, this->img_cols, this->img_rows,
								kernel_size, sigma_spatial2_inv_half, sigma_color2_inv_half);
	//hipDeviceSynchronize();

	hipMemcpy(host_dst_img, dev_result_img,
		this->img_size * sizeof(unsigned char), hipMemcpyDeviceToHost);

}

